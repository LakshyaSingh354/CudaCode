#include <hip/hip_runtime.h>
#include <iostream>

#define TILE_SIZE 2

__global__ void matmul_naive(float* A, float* B, float* C, int m, int k, int n){
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    if(row < m && col < n){
        float sum = 0.0f;
        for (int p = 0; p < k; ++p){
            sum += A[row * k + p] * B[p * n + col];
        }
        C[row * n + col] = sum;
    }
}

__global__ void matmul_tiled(float* A, float* B, float* C, int m, int k, int n){
    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;

    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];

    float sum = 0.0f;
    for(int t = 0; t < (k + TILE_SIZE - 1)/TILE_SIZE; ++t){
        int Arow = row;
        int Acol = t * TILE_SIZE + threadIdx.x;
        int Brow = t * TILE_SIZE + threadIdx.y;
        int Bcol = col;

        if (Arow < m && Acol < k){
            As[threadIdx.y][threadIdx.x] = A[Arow * k + Acol];
        } else{
            As[threadIdx.y][threadIdx.x] = 0.0f;
        }

        if (Brow < k && Bcol < n){
            Bs[threadIdx.y][threadIdx.x] = B[Brow * n + Bcol];
        } else{
            Bs[threadIdx.y][threadIdx.x] = 0.0f;
        }

        __syncthreads();

        for(int p = 0; p < k; p++){
            sum += As[threadIdx.y][p] * Bs[p][threadIdx.x];
        }
        __syncthreads();
    }

    if(row < m && col < n){
        C[row * n + col] = sum;
    }

}

int main(){
    int m = 4, k = 4, n = 4;
    size_t size_A = m * k * sizeof(float);
    size_t size_B = k * n * sizeof(float);
    size_t size_C = m * n * sizeof(float);


    float* h_A = (float*)malloc(size_A);
    float* h_B = (float*)malloc(size_B);
    float* h_C = (float*)malloc(size_C);

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size_A);
    hipMalloc(&d_B, size_B);
    hipMalloc(&d_C, size_C);

    for (int i = 0; i < m; ++i){
        for(int j = 0; j < k; ++j){
            h_A[i*k + j] = i*0.1 + j*0.2;
            // h_A[i*k + j] = 1.0;
        }
    }
    for (int i = 0; i < k; ++i){
        for(int j = 0; j < n; ++j){
            h_B[i*n + j] = i*0.2 + j*0.1;
            // h_B[i*n + j] = 2.0;
        }
    }
    printf("A = [");
    for (int i = 0; i < m; ++i){
        printf("[ ");
        for(int j = 0; j < k; ++j){
            printf("%.2f ", h_A[i*k + j]);
        }
        printf("]\n");
    }

    printf("]\n");
    printf("B = [");
    for (int i = 0; i < m; ++i){
        printf("[ ");
        for(int j = 0; j < k; ++j){
            printf("%.2f ", h_B[i*n + j]);
        }
        printf("]\n");
    }
    printf("]\n");

    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
    dim3 numBlocks((n + TILE_SIZE - 1)/TILE_SIZE, (m + TILE_SIZE - 1)/TILE_SIZE);

    matmul_tiled<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, m, k, n);

    hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);

    printf("C = [");
    for (int i = 0; i < m; ++i){
        printf("[ ");
        for(int j = 0; j < k; ++j){
            printf("%.2f ", h_C[i*n + j]);
        }
        printf("]\n");
    }
    printf("]\n");

    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    free(h_A); free(h_B); free(h_C);
}