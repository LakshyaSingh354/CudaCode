#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>

#define CUDA_CHECK(call) \
    { hipError_t err = call; if (err != hipSuccess) { \
        std::cerr << "CUDA error: " << hipGetErrorString(err) << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
        exit(1); }}

__global__ void const_vec(float* arr, int N){
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < N){
        arr[i] = 1.0f;
    }
}

int main(){
    int N = 16;

    size_t size = N * sizeof(float);
    std::vector<float> arr_h(N);

    float* arr_d;
    CUDA_CHECK(hipMalloc(&arr_d, size));

    int threadsPerBlock = 256;
    int numBlocks = (N + threadsPerBlock - 1)/threadsPerBlock;

    const_vec<<<numBlocks, threadsPerBlock>>>(arr_d, N);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(arr_h.data(), arr_d, size, hipMemcpyDeviceToHost));

    for(int i = 0; i < N; ++i){
        std::cout << arr_h[i] << " ";
    }
    std:: cout << std::endl;

    CUDA_CHECK(hipFree(arr_d));

    return 0;
}