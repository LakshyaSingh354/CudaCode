#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>

#define CUDA_CHECK(call) \
    { hipError_t err = call; if (err != hipSuccess) { \
        std::cerr << "CUDA error: " << hipGetErrorString(err) << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
        exit(1); }}

__global__ void vec_scaling(float* arr, int N, int c){
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < N){
        arr[i] *= c;
    }
}

int main(){
    int N = 16;
    size_t size = N * sizeof(float);
    int c = 2;

    std::vector<float> arr_h(N, 1.5f);

    std::cout << "Original Vector: [ ";
    for(int i = 0; i < N; ++i){
        std::cout << arr_h[i] << " ";
    }
    std::cout << "]" << std::endl;

    float* arr_d;

    CUDA_CHECK(hipMalloc(&arr_d, size));
    CUDA_CHECK(hipMemcpy(arr_d, arr_h.data(), size, hipMemcpyHostToDevice));

    int threadsPerBlock = 256;
    int numBlocks = (N + threadsPerBlock - 1) / threadsPerBlock;

    vec_scaling<<<numBlocks, threadsPerBlock>>>(arr_d, N, c);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(arr_h.data(), arr_d, size, hipMemcpyDeviceToHost));

    std::cout << "Scaled Vector: [ ";
    for(int i = 0; i < N; ++i){
        std::cout << arr_h[i] << " ";
    }
    std::cout << "]" << std::endl;

    CUDA_CHECK(hipFree(arr_d));

    return 0;
}