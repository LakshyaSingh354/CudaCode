#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>

#define CUDA_CHECK(call) \
    { hipError_t err = call; if (err != hipSuccess) { \
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
        exit(1); }}

__global__ void vec_add(float* A, float* B, float* C, int N){
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if(i < N){
        C[i] = A[i] + B[i];
    }
}

int main(){
    int N = 16;
    size_t size = N * sizeof(float);

    std::vector<float> h_A(N, 1.0f), h_B(N, 2.0f), h_C(N);

    std::cout << "A = [ ";
    for (int i = 0; i < N; ++i){
        std::cout << h_A[i] << " ";
    }
    std::cout << "]" << std::endl;

    std::cout << "B = [ ";
    for (int i = 0; i < N; ++i){
        std::cout << h_B[i] << " ";
    }
    std::cout << "]" << std::endl;

    float *d_A, *d_B, *d_C;
    CUDA_CHECK(hipMalloc(&d_A, size));
    CUDA_CHECK(hipMalloc(&d_B, size));
    CUDA_CHECK(hipMalloc(&d_C, size));

    CUDA_CHECK(hipMemcpy(d_A, h_A.data(), size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, h_B.data(), size, hipMemcpyHostToDevice));

    int threadsPerBlock = 256;
    int numBlocks = (N + threadsPerBlock - 1) / threadsPerBlock;

    vec_add<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, N);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(h_C.data(), d_C, size, hipMemcpyDeviceToHost));

    std::cout << "A + B = [ ";
    for (int i = 0; i < N; ++i){
        std::cout << h_C[i] << " ";
    }
    std::cout << "]" << std::endl;

    CUDA_CHECK(hipFree(d_A)); CUDA_CHECK(hipFree(d_B)); CUDA_CHECK(hipFree(d_C));

    return 0;
}